#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <fstream>
#include <vector>
#include <random>
#include <string>
#include <sstream>
#include <iomanip>
#include "hip/hip_runtime.h"

using namespace std;


void cudaCheck(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        cerr << "CUDA error during " << context << ": "
            << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }
}


__global__ void computeMatrixProduct(const int* matA, const int* matB, int* matResult, int dim) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < dim && col < dim) {
        int partial = 0;
        for (int k = 0; k < dim; ++k) {
            partial += matA[row * dim + k] * matB[k * dim + col];
        }
        matResult[row * dim + col] = partial;
    }
}


int randInRange(int low, int high) {
    static mt19937 rng(random_device{}());
    uniform_int_distribution<int> dist(low, high);
    return dist(rng);
}


vector<vector<int>> createRandMatrix(int size, int min = 1, int max = 100) {
    vector<vector<int>> matrix(size, vector<int>(size));
    for (int i = 0; i < size; ++i)
        for (int j = 0; j < size; ++j)
            matrix[i][j] = randInRange(min, max);
    return matrix;
}


void storeMatrix(const vector<vector<int>>& matrix, const string& filename) {
    ofstream outfile(filename);
    for (const auto& row : matrix) {
        for (size_t i = 0; i < row.size(); ++i) {
            outfile << row[i];
            if (i < row.size() - 1) outfile << ",";
        }
        outfile << "\n";
    }
}


float measureKernelExecutionTime(const int* d_A, const int* d_B, int* d_Result, int n) {
    dim3 blockSize(8, 8);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x,
        (n + blockSize.y - 1) / blockSize.y);

    hipEvent_t start, stop;
    cudaCheck(hipEventCreate(&start), "event create start");
    cudaCheck(hipEventCreate(&stop), "event create stop");

    cudaCheck(hipEventRecord(start, 0), "event record start");
    computeMatrixProduct << <gridSize, blockSize >> > (d_A, d_B, d_Result, n);
    cudaCheck(hipGetLastError(), "kernel execution");
    cudaCheck(hipEventRecord(stop, 0), "event record stop");

    cudaCheck(hipEventSynchronize(stop), "event synchronize stop");

    float milliseconds = 0;
    cudaCheck(hipEventElapsedTime(&milliseconds, start, stop), "event elapsed time");

    cudaCheck(hipEventDestroy(start), "event destroy start");
    cudaCheck(hipEventDestroy(stop), "event destroy stop");

    return milliseconds;
}


float gpuMatrixMultiply(const vector<vector<int>>& A, const vector<vector<int>>& B, vector<vector<int>>& result) {
    int n = A.size();

    vector<int> flatA(n * n), flatB(n * n), flatResult(n * n);
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < n; ++j) {
            flatA[i * n + j] = A[i][j];
            flatB[i * n + j] = B[i][j];
        }

    int* d_A, * d_B, * d_Result;
    size_t bytes = n * n * sizeof(int);

    cudaCheck(hipMalloc(&d_A, bytes), "malloc A");
    cudaCheck(hipMalloc(&d_B, bytes), "malloc B");
    cudaCheck(hipMalloc(&d_Result, bytes), "malloc Result");

    cudaCheck(hipMemcpy(d_A, flatA.data(), bytes, hipMemcpyHostToDevice), "copy A");
    cudaCheck(hipMemcpy(d_B, flatB.data(), bytes, hipMemcpyHostToDevice), "copy B");

    float milliseconds = measureKernelExecutionTime(d_A, d_B, d_Result, n);

    cudaCheck(hipMemcpy(flatResult.data(), d_Result, bytes, hipMemcpyDeviceToHost), "copy Result");

    cudaCheck(hipFree(d_A), "free A");
    cudaCheck(hipFree(d_B), "free B");
    cudaCheck(hipFree(d_Result), "free Result");

    result.resize(n, vector<int>(n));
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < n; ++j)
            result[i][j] = flatResult[i * n + j];

    return milliseconds;
}


int main() {
    int deviceCount = 0;
    cudaCheck(hipGetDeviceCount(&deviceCount), "get device count");
    if (deviceCount == 0) {
        cerr << "CUDA-capable devices not found" << endl;
        return 1;
    }
    cout << "Detected CUDA devices: " << deviceCount << endl;

    const string outputDir = "C:/Users/Sofia/Desktop/CudaRuntime1/CudaRuntime1.sln/results";
    ofstream perfData("gpu_performance.txt");

    for (int dim = 250; dim <= 3000; dim += 250) {
        cout << "Matrix dimension: " << dim << "x" << dim << endl;

        auto matA = createRandMatrix(dim);
        auto matB = createRandMatrix(dim);

        string fileA = outputDir + "/matrix_" + to_string(dim) + "_A.csv";
        string fileB = outputDir + "/matrix_" + to_string(dim) + "_B.csv";
        storeMatrix(matA, fileA);
        storeMatrix(matB, fileB);

        vector<vector<int>> product;
        float kernelTime = gpuMatrixMultiply(matA, matB, product);

        string resultFile = outputDir + "/gpu_result_" + to_string(dim) + ".csv";
        storeMatrix(product, resultFile);

        perfData << dim << "\t" << fixed << setprecision(3) << kernelTime << endl;
        cout << "  Kernel time: " << fixed << setprecision(3) << kernelTime << " ms" << endl;
    }

    perfData.close();
    cout << "Performance data saved to gpu_performance.txt" << endl;
    return 0;
}
